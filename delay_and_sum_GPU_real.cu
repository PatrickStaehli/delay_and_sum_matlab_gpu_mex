#include "hip/hip_runtime.h"
 /*
 * delay_and_sum.cu -  MATLAB external interfaces GPU implementation of a delay and sum algorithm for plane-wave acquisitions.
 * Reconstructs a set of plane-wave ultrasound acquisitions with various different acquistition angles. 
 * 
 *
 *	Input:  	raw_data 				- 3D Matlab GPU array (axial, lateral, acquisition angle)
 * 				acquisition_parameter	- Struct 
 *											- F 				Sampling frequency [MHz]			
 *											- pitch 			Element Pitch [mm] 					
 * 											- c0 				Acquisition speed-of-sound [mm/us] 	
 *				recon_parameter			- Struct
 *											- lense.thickness	Thickness of the acoustic transducer lense
 *											- lense.c 			Speed-of-sound of the acoustic transducer lense
 *											- c 				Anticipated speed-of-sound for delay-and-sum
 *											- apodtan			Tan of the apodization angle
 * 				tx_delays 				- 1D Matlab GPU array of transmit delays for each steering angle
 * 				tx_angles				- 1D Matlab GPU array of steering angles [rad]
 * 
 * 	Output:		rekon 					- 3D Matlab GPU array of the delay-and-summed input signal (axial, lateral, acquisition angle)
 *
 *
 *
 * The calling syntax from Matlab is:
 *
 *		outMatrix = delay_and_sum_GPU(raw_data,acquisition_parameter,recon_parameter, tx_delays, tx_angles)
 *
 * This is a MEX file for MATLAB.
 * 
*/



#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <math.h>


/* Create parameter Struct */
struct parameters {
    double sampling_frequency;
	double pitch;
	double speed_of_sound;
	double steering_delay;
	double lense_thickness;
	double lense_speed_of_sound;
	double apodtan;
	double tx_angle;
    int time_steps;
    int number_of_elements;
	int num_threads;
};

/* Input error handling */
/*--------------------------------------------------------------------------------------------------*/
void inputErrorHandling(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){
    /* Verify if the inputs have a valid shape and that all requred inputs are given.
     *
     * Expected input from matlab: (complex Signal, acquisition_parameter, recon_parameters)
     *
     * Expected output to matlab: complex image 
     *
    */
  
    if(nrhs != 5)
      mexErrMsgIdAndTxt( "MATLAB:convec:invalidNumInputs",
              "Five inputs required.");
    if(nlhs > 2)
      mexErrMsgIdAndTxt( "MATLAB:convec:maxlhs",
              "Only one output argument of size T*X allowed");
	
	// Check if the input is GPU Array
	if (!(mxIsGPUArray(prhs[0]))) {
        mexErrMsgIdAndTxt("parallel:gpu:mexGPUExample:InvalidInput", "Invalid input to MEX file.");
    }
    // Check that the acquisition_parameter structure contains all requred fields
    if(mxGetPr(mxGetField(prhs[1],0,"pitch")) == NULL || mxGetPr(mxGetField(prhs[1],0,"F")) == NULL || mxGetPr(mxGetField(prhs[1],0,"c0")) == NULL){
        mexErrMsgIdAndTxt( "MATLAB:convec:inputsNotComplex",
               " Requred input: (complex_signal, acquisition_parameter, recon_parameter, tx_angle, tx_delay), with  \n" 
               " acquisition_parameter.F \t \t -> sampling frequency [MHz] \n"
               " acquisition_parameter.pitch \t -> element pitch  [mm] \n"
               " acquisition_parameter.c0 \t \t -> Speed-of-sound used for the delay calculation [mm/us]");
    }
    // Check that the recon_parameter structure contains all requred fields
    if(mxGetPr(mxGetField(prhs[2],0,"c")) == NULL || mxGetPr(mxGetField(prhs[2],0,"apodtan")) == NULL || mxGetPr(mxGetField(mxGetField(prhs[2],0,"lense"),0,"thickness")) == NULL ||  mxGetPr(mxGetField(mxGetField(prhs[2],0,"lense"),0,"c")) == NULL){
        mexErrMsgIdAndTxt( "MATLAB:convec:inputsNotComplex",
               " Requred input: (complex_signal, acquisition_parameter, recon_parameter, tx_angle, tx_delay), with  \n" 
               " recon_parameter.lense.thickness \t -> Thickness of the transducer lense [mm] \n"
               " recon_parameter.lense.c \t \t \t -> Speed-of-sound of the transducer lense [mm/us] \n"
               " recon_parameter.c \t \t \t \t \t -> anticipated speed-of-sound [mm/us] \n"
               " recon_parameter.apodtan \t \t \t -> Tan of the apodization angle \n");
    }
    
}


//* Device code */
/*--------------------------------------------------------------------------------------------------*/
void __global__ delay_and_sum(double const * const raw_data, 
                         double * const rekon,
						 double const * const delay_F, double const * const tx_angles,
						 struct parameters recon_parameters,
                         int const N)
{
	// Calculate the global linear index, assuming a 1-d grid.
    int const iter = blockDim.x * blockIdx.x + threadIdx.x;
	
	// variable declaration
	//thrust::complex<double> phasefac (0,1);
	//double const pi = 3.14159265359;
	//thrust::complex<double> j(0,1);
	
	double tx_angle;		// Current Tx angle
	
	double pitch_F;  		// Element pitch in units of sampling_frequency/SoS
	double tx_delay;		// Delay from element to pixel
	
	double tx_lense_cos; 	// cosine of the lense tx path
	double tx_correction;	// Lense delay correction  
	
	double z_T;				// To be reconstructed depth in units of sampling_frequency/SoS

	double roundtrip_tx = 0;	// Total roundtrip time in TX (steering delay + lense delay + transmit delay)
    
	double delata_pos_rx; 	// Difference of the Rx element position from the pixel positoin
	
	double rx_delay;		// Delay form the pixel to the Rx element
	double roundtrip_rx = 0;	// Total roundtrip time in RX
	
	double rx_element_weight; // Weight of the rx element in dependence of the element rx angle
	
	double rx_angle_sin;	// Sinus of the rx element angle
	double rx_lense_sin;	// Sinus of propagation angle inside lense 
	double rx_lense_cos;	// Cosine of the propagation angle inside the lense
	double rx_correction;	// Lense delay correction in Rx
	
	double total_roundtrip_time; // Total roundtrip time consisting of Tx delay and Rx delay
	
	double rekon_signal = 0;  // Reconstructed signal
	
	// To be reconstructed pixel (z,x,angle)
	int z = iter%recon_parameters.time_steps+ 1;
	int x = (iter / recon_parameters.time_steps) % recon_parameters.number_of_elements	+ 1;
	int angle_num = iter/(recon_parameters.time_steps*recon_parameters.number_of_elements); 
	
	// The tx_angle of the kernel 
	tx_angle = tx_angles[angle_num];

	
	if (iter < N) {	// must be a valid index
		
		pitch_F = (recon_parameters.pitch / recon_parameters.speed_of_sound) * recon_parameters.sampling_frequency; // Element pitch [F/c]
		z_T = (z-1.0) / 2.0;	 // current depth in numbers of speed_of_sound/sampling_frequency
		
		// Tx delay
		tx_delay = (-roundf(recon_parameters.number_of_elements / 2) + (double)x) * pitch_F * sinf(tx_angle) ; 	// Calculate the delay [F/c]
		// Tx delay due to acoustic lense of the transducer
		tx_lense_cos = sqrtf(1 - powf(sinf(abs(tx_angle)),2) * powf(recon_parameters.lense_speed_of_sound,2) / powf(recon_parameters.speed_of_sound,2)); 
		tx_correction = recon_parameters.lense_thickness / recon_parameters.lense_speed_of_sound * tx_lense_cos * recon_parameters.sampling_frequency; // Lense delay correction [F/c]
		
		// Total Tx time
		roundtrip_tx = tx_delay + tx_correction + z_T*cosf(tx_angle);

		
		// Rx Delay
		for (int i_Rx_Element=0; i_Rx_Element< recon_parameters.number_of_elements; i_Rx_Element++ ){  // Rx element offset from insonified pixel
			
			delata_pos_rx = abs((x-1)-i_Rx_Element)*pitch_F; // Difference of the element from the to-be reconsturcted pixel in x
			rx_delay = sqrtf(powf(delata_pos_rx,2) + powf(z_T,2));
			
			rx_element_weight = expf(-powf(delata_pos_rx,2)/powf(z_T,2)/powf(recon_parameters.apodtan,2)); // Weight of the receive element in depencence of the Rx angle
			
			if (recon_parameters.apodtan == 0) { // If the weight is 0, equal weighting for all elementes
				rx_element_weight = 1.0;
			}
			
			
			if (rx_element_weight>1e-3) { // We exclude elements with a small weight
				
				//Rx lense correction	
				rx_angle_sin = delata_pos_rx / rx_delay; // sinus of receive angles 
				rx_lense_sin = rx_angle_sin * recon_parameters.lense_speed_of_sound / recon_parameters.speed_of_sound; // sinus of propagation angle inside lense 
				rx_lense_cos = sqrtf(1-powf(rx_lense_sin,2)); // cosine of propagation angle inside lense 
				rx_correction = recon_parameters.lense_thickness * rx_lense_cos * recon_parameters.sampling_frequency / recon_parameters.lense_speed_of_sound; // delay correction in numbers of lense_speed_of_sound/sampling_frequency
				roundtrip_rx = rx_delay + rx_correction;	
				
				total_roundtrip_time =  roundtrip_rx - delay_F[angle_num] + roundtrip_tx;
				
				
				if (total_roundtrip_time >= 1 && total_roundtrip_time <= recon_parameters.time_steps){ // only valid indices
					
					// Determine the 1d Index of the signal position
					long idx_angle = recon_parameters.time_steps*recon_parameters.number_of_elements*(angle_num);
					long idx_sig = (round(total_roundtrip_time-1)) + (i_Rx_Element)*(recon_parameters.time_steps) + idx_angle;
					
					// Add the signal to the reconstructed signal
					rekon_signal +=  raw_data[idx_sig] * rx_element_weight;
				}
			}
		}
		rekon[iter] = rekon_signal;
		
    }
}




/* Host code */
/*--------------------------------------------------------------------------------------------------*/
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
	
	/* Variable declarations */
	/*--------------------------------------------------------------------------------------------------*/
	
    // GPU Variables
	mxGPUArray const *raw_data, *delay, *tx_angle;
    mxGPUArray *rekon;
    double const *raw_dataP, *delayP, *tx_angleP;
    double *rekonP;
    int N;

	// Choose a reasonably sized number of threads for the block. 
    int const threadsPerBlock = 256*2;
    int blocksPerGrid;
	
	
	/* Initialize the MathWorks GPU API. */
	/*--------------------------------------------------------------------------------------------------*/
	
    mxInitGPU();
    // Input Errro handling
    inputErrorHandling(nlhs, plhs, nrhs, prhs);
  
    
	/* Read the inputs from Matlab */
	/*--------------------------------------------------------------------------------------------------*/
	raw_data = 	mxGPUCreateFromMxArray(prhs[0]);
    raw_dataP = 		(double const *)(mxGPUGetDataReadOnly(raw_data)); // Pointer to raw_data
	
	delay = 	mxGPUCreateFromMxArray(prhs[3]);
	tx_angle = 	mxGPUCreateFromMxArray(prhs[4]);
	delayP =	(double const *)(mxGPUGetDataReadOnly(delay)); // Pointer to delay
	tx_angleP =	(double const *)(mxGPUGetDataReadOnly(tx_angle)); // Pointer to tx_angle
	
	
	// Create a GPUArray to hold the result and get its underlying pointer.
    rekon = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(raw_data),
                            mxGPUGetDimensions(raw_data),
                            mxGPUGetClassID(raw_data),
                            mxGPUGetComplexity(raw_data),
                            MX_GPU_DO_NOT_INITIALIZE);
    rekonP = (double *)(mxGPUGetData(rekon)); // Pointer to rekon
	
	
	// Get the dimension of the input array
	const mwSize *dim_array;
    dim_array = mxGPUGetDimensions(raw_data);
    
	// Read the parameters and store them in a struct
	struct parameters recon_parameters;
 
    recon_parameters.time_steps = dim_array[0];
    recon_parameters.number_of_elements = dim_array[1];
    recon_parameters.sampling_frequency = *mxGetPr(mxGetField(prhs[1],0,"F"));
	recon_parameters.pitch = *mxGetPr(mxGetField(prhs[1],0,"pitch"));
    recon_parameters.speed_of_sound = *mxGetPr(mxGetField(prhs[2],0,"c"));
    recon_parameters.steering_delay = *mxGetPr(prhs[3]);
    recon_parameters.lense_thickness = *mxGetPr(mxGetField(mxGetField(prhs[2],0,"lense"),0,"thickness"));
    recon_parameters.lense_speed_of_sound = *mxGetPr(mxGetField(mxGetField(prhs[2],0,"lense"),0,"c"));
    recon_parameters.apodtan = *mxGetPr(mxGetField(prhs[2],0,"apodtan"));
    recon_parameters.tx_angle = *mxGetPr(prhs[4]);
    
	
	/* Execute device code */
	/*--------------------------------------------------------------------------------------------------*/
	
	N = (int)(mxGPUGetNumberOfElements(raw_data));
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	delay_and_sum<<<blocksPerGrid, threadsPerBlock>>>(raw_dataP, rekonP, delayP, tx_angleP, recon_parameters, N);	
	
	
    /* Create Output to Matlab */
	/*--------------------------------------------------------------------------------------------------*/
	
	//Wrap the result up as a MATLAB gpuArray for return.
    plhs[0] = mxGPUCreateMxArrayOnGPU(rekon);
	
	
	/* Clearing GPU memory cache */
	/*--------------------------------------------------------------------------------------------------*/
	mxGPUDestroyGPUArray(raw_data);
	mxGPUDestroyGPUArray(rekon);
	mxGPUDestroyGPUArray(delay);
    mxGPUDestroyGPUArray(tx_angle);
	
}
